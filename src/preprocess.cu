#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

#include "glog/logging.h"
#include "detect.hpp"
#include "config.hpp"
#include "common.hpp"

using namespace google;

__device__ void affine_project(float *mat, int x, int y, float *proj_x, float *proj_y)
{
    // matrix
    // m0, m1, m2
    // m3, m4, m5
    *proj_x = mat[0] * x + mat[1] * y + mat[2];
    *proj_y = mat[3] * x + mat[4] * y + mat[5];
}

// warp affine transformation by bilinear interpolation
__global__ void warp_affine(
    uint8_t *src, int src_line_size, int src_width, int src_height,
    uint8_t *dst, int dst_line_size, int dst_width, int dst_height,
    uint8_t fill_value, AffineMatrix M)
{
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;

    if (dx >= dst_width || dy >= dst_height)
        return;

    float c0 = fill_value, c1 = fill_value, c2 = fill_value;
    float src_x = 0;
    float src_y = 0;
    affine_project(M.inv_mat, dx, dy, &src_x, &src_y);

    if (src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height)
    {
        // out of range
        // when src_x < -1，high_x < 0，out of range
        // when src_x >= -1，high_x >= 0，in range
    }
    else
    {
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values[] = {fill_value, fill_value, fill_value};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t *v1 = const_values;
        uint8_t *v2 = const_values;
        uint8_t *v3 = const_values;
        uint8_t *v4 = const_values;
        if (y_low >= 0)
        {
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }

        if (y_high < src_height)
        {
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }

        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
    }

    uint8_t *pdst = dst + dy * dst_line_size + dx * 3;
    pdst[0] = c0;
    pdst[1] = c1;
    pdst[2] = c2;
}

void Detect::preprocess(std::vector<cv::Mat> &images)
{
   for (auto &src : images)
   {
        uint8_t* d_ptr_src; // device pointer for src image
        uint8_t* d_pre_dst; // device pointer for dst image
        int src_w = src.cols; // src image width
        int src_h = src.rows; // src image height
        int dst_w = this->input_width; // dst image width
        int dst_h = this->input_height; // dst image height
        size_t src_size = src_w * src_h * 3 * sizeof(uint8_t); // src image size
        size_t dst_size = dst_w * dst_h * 3 * sizeof(uint8_t); // dst image size

        CUDA_CHECK(hipMalloc((uint8_t **)&d_ptr_src, src_size)); 
        CUDA_CHECK(hipMalloc((uint8_t **)&d_pre_dst, dst_size));
        CUDA_CHECK(hipMemcpy(d_ptr_src, src.data, src_size, hipMemcpyHostToDevice));
        
        // compute affine tranformation matrix
        (this->affine_matrix).compute(cv::Size(src_w, src_h), cv::Size(dst_w, dst_h));

        dim3 block(32, 32);
        dim3 grid((dst_w + block.x - 1) / block.x, (dst_h + block.y - 1) / block.y);

        LOG(INFO) << "warp_affine kernel launch with " 
            << grid.x << "x" << grid.y << " blocks of " 
            << block.x << "x" << block.y << " threads";

        // do letterbox transformation on src image
        // src: [src_h, src_w, 3], dst: [dst_h, dst_w, 3]
        warp_affine<<<grid, block, 0, nullptr>>>(
            d_ptr_src, src_w * 3, src_w, src_h,
            d_pre_dst, dst_w * 3, dst_w, dst_h,
            114, this->affine_matrix
        );
        
   }
}